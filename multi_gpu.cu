#include "hip/hip_runtime.h"
#include <stdio.h>

#define MIN_GLOBAL_MEMORY 4LL * 1024 * 1024 * 1024 // 4 GB in bytes

// CUDA kernel function (simple example)
__global__ void simpleKernel() {
    printf("Hello from the GPU!\n");
}

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    printf("Number of GPU Devices: %d\n", nDevices);

    int currentChosenDeviceNumber = -1;

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device Compute Major: %d Minor: %d\n", prop.major, prop.minor);
        printf("  Max Thread Dimensions: [%d][%d][%d]\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
        printf("  Number of Multiprocessors: %d\n", prop.multiProcessorCount);
        printf("  Device Clock Rate (KHz): %d\n", prop.clockRate);
        printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
        printf("  Registers Per Block: %d\n", prop.regsPerBlock);
        printf("  Registers Per Multiprocessor: %d\n", prop.regsPerMultiprocessor);
        printf("  Shared Memory Per Block: %zu\n", prop.sharedMemPerBlock);
        printf("  Shared Memory Per Multiprocessor: %zu\n", prop.sharedMemPerMultiprocessor);
        printf("  Total Constant Memory (bytes): %zu\n", prop.totalConstMem);
        printf("  Total Global Memory (bytes): %zu\n", prop.totalGlobalMem);
        printf("  Warp Size: %d\n", prop.warpSize);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);

        // Check if the device meets the minimum global memory requirement
        if (prop.totalGlobalMem >= MIN_GLOBAL_MEMORY) {
            currentChosenDeviceNumber = i;
            break;
        }
    }

    // Print out the chosen device
    printf("The chosen GPU device has an index of: %d\n", currentChosenDeviceNumber);

    if (currentChosenDeviceNumber != -1) {
        // Set the chosen device
        hipSetDevice(currentChosenDeviceNumber);

        // Execute a simple kernel on the chosen device
        simpleKernel<<<1, 1>>>();
        hipDeviceSynchronize();  // Ensure the kernel execution is completed
    } else {
        printf("No suitable GPU device found with the required minimum global memory.\n");
    }

    return 0;
}
